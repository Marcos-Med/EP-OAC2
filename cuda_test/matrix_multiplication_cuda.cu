// Filename: matrix_multiplication_cuda.cu

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 3  // Dimension of the matrices (n x n)

// CUDA kernel for matrix multiplication
__global__ void matrixMultiplyKernel(double *A, double *B, double *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Row index of C and A
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Column index of C and B

    if (row < n && col < n) {
        double value = 0.0;
        for (int k = 0; k < n; k++) {
            value += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = value;
    }
}

// Function to print the matrix
void printMatrix(double *matrix, int n) {
    for (int i = 0; i < n * n; i++) {
        printf("%.1f ", matrix[i]);
        if ((i + 1) % n == 0) {
            printf("\n");
        }
    }
}

int main() {
    int n = N;
    size_t size = n * n * sizeof(double);

    // Allocate memory on the host (CPU)
    double *h_A = (double *)malloc(size);
    double *h_B = (double *)malloc(size);
    double *h_C = (double *)malloc(size);

    // Initialize matrices with 2.0
    for (int i = 0; i < n * n; i++) {
        h_A[i] = 2.0;
        h_B[i] = 2.0;
        h_C[i] = 0.0;
    }

    // Allocate memory on the device (GPU)
    double *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // Launch the matrix multiplication kernel on the GPU
    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    // Record the stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate the elapsed time in milliseconds
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel execution time: %.3f ms\n", elapsedTime);

    // Copy result matrix from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the result matrix
    printf("Result matrix C (after multiplication):\n");
    printMatrix(h_C, n);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Reset the device
    hipDeviceReset();

    return 0;
}